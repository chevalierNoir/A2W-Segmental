// cuda_error_checking.h
#include "cuda_error_checking.h"
#include <stdio.h>
 
// Assumes single device when calling hipDeviceReset(); and exit(code);
// In some cases a more lengthy program clean up / termination may be needed
 
void checkError(hipError_t code, char const * func, const char *file, const int line, bool abort)
{
    if (code != hipSuccess) 
    {
        const char * errorMessage = hipGetErrorString(code);
        fprintf(stderr, "CUDA error returned from \"%s\" at %s:%d, Error code: %d (%s)\n", func, file, line, code, errorMessage);
        if (abort){
            hipDeviceReset();
            exit(code);
        }
    }
    else if (PRINT_ON_SUCCESS)
    {
        const char * errorMessage = hipGetErrorString(code);
        fprintf(stderr, "CUDA error returned from \"%s\" at %s:%d, Error code: %d (%s)\n", func, file, line, code, errorMessage);
    }
}
 
void checkLastError(char const * func, const char *file, const int line, bool abort)
{
    hipError_t code = hipGetLastError();
    if (code != hipSuccess)
    {
        const char * errorMessage = hipGetErrorString(code);
        fprintf(stderr, "CUDA error returned from \"%s\" at %s:%d, Error code: %d (%s)\n", func, file, line, code, errorMessage);
        if (abort) {
            hipDeviceReset();
            exit(code);
        }
    }
    else if (PRINT_ON_SUCCESS)
    {
        const char * errorMessage = hipGetErrorString(code);
        fprintf(stderr, "CUDA error returned from \"%s\" at %s:%d, Error code: %d (%s)\n", func, file, line, code, errorMessage);
    }
}
